#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#define N 32764  // Slightly adjusted for better memory alignment
#define ITERS 1024
#define THREADS_PER_BLOCK 256
#define BLOCKS ((N * N) / (ITERS * THREADS_PER_BLOCK))
__global__ void fma_kernel(float *data, int matrix_size, float mul, float add) {
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned stride = (tid * 7919) % 1023 + 1;  // Prime-based stride variation
    unsigned start = tid * 12345;  // Arbitrary starting offset
    
    for (int i = 0; i < ITERS; i++) {
        unsigned idx = (start + i * stride) % matrix_size;
        float val = data[idx];
        data[idx] = fmaf(val, mul, add);
    }
}
int main() {
    const size_t size = N * N * sizeof(float);
    const int matrix_size = N * N;
    float *h_data, *d_data;
    // Allocate pinned host memory
    hipHostMalloc(&h_data, size, hipHostMallocDefault);
    // Allocate device memory
    hipMalloc(&d_data, size);
    // Initialize host array with sample data
    for (int i = 0; i < matrix_size; i++) {
        h_data[i] = 1.0f;
    }
    // Copy data to device
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    // Launch kernel with error checking
    dim3 grid(BLOCKS);
    dim3 block(THREADS_PER_BLOCK);
    fma_kernel<<<grid, block>>>(d_data, matrix_size, 2.0f, 3.0f);
    hipGetLastError();
    hipDeviceSynchronize();
    // Copy data back to host
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    // Cleanup
    hipFree(d_data);
    hipHostFree(h_data);
    return 0;
}